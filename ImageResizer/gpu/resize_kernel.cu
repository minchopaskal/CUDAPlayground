// Includes that fix syntax highlighting
#ifdef IMG_RESIZER_DEBUG
#include "device_launch_parameters.h"
#include "stdio.h"
#include "math_functions.h"
#endif


#include <hip/hip_runtime.h>
#include "hip/hip_math_constants.h"

#define gvoid  __global__ void
#define gfloat __global__ float
#define gint   __global__ int

#define dvoid  __device__ void
#define dfloat __device__ float
#define dint   __device__ int

#define cvoid  __constant__ void
#define cfloat __constant__ float
#define cint   __constant__ int

typedef float (*samplingKernel)(float x, float y, int window);

extern "C" {

	cint arrSize;
	gvoid adder(int *arrA, int *arrB, int *result) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		idx = min(idx, arrSize - 1);
		result[idx] = arrA[idx] + arrB[idx];
	}

	dfloat sinc(float x) {
		float PI_x = HIP_PI_F * x;
		return sin(PI_x) / (PI_x);
	}

	dfloat lanczos2(float x) {
		if (x > -1e-6f && x < 1e-6f) {
			return 1.f;
		}

		if (x < -2.f || x > 2.f) {
			return 0.f;
		}

		return sinc(x) * sinc(x / 2.f);
	}

	dfloat lanczos3(float x) {
		if (x > -1e-6f && x < 1e-6f) {
			return 1.f;
		}

		if (x < -3.f || x > 3.f) {
			return 0.f;
		}

		return sinc(x) * sinc(x / 3.f);
	}

	dfloat lanczos2D(float x, float y, int window) {
		if (window != 2 && window != 3) {
			return 0.f;
		}

		if (window == 2) {
			return lanczos2(x) * lanczos2(y);
		}
		
		return lanczos3(x) * lanczos3(y);
	}

	dfloat nearestNeighbour(float x, float y, int window) {
		return x >= -0.5f && x <= 0.5f && y >= -0.5f && y <= 0.5f;
	}

	dvoid convolve(
		const unsigned char *inImg,
		samplingKernel kernel,
		float2 sample,
		int2 rangeX,
		int2 rangeY,
		int inputWidth,
		int numComp,
		int window,
		unsigned char *result
	) {
		float result_[4];
		for (int i = 0; i < numComp; ++i) {
			result_[i] = 0;
		}

		for (int i = rangeY.x; i < rangeY.y; ++i) {
			for (int j = rangeX.x; j < rangeX.y; ++j) {
				int inputIdx = (i * inputWidth + j) * numComp;
				float kernelValue = kernel(sample.x - j, sample.y - i, window);

				for (int k = 0; k < numComp; ++k) {
					const float sampleWeighted = float(inImg[inputIdx + k]) * kernelValue;
					result_[k] += sampleWeighted;
				}
			}
		}

		for (int i = 0; i < numComp; ++i) {
			result[i] = (unsigned char)(min(max(0.f, result_[i]), 255.f));
		}
	}

	// TODO: put params in a struct and make it a constant variable
	gvoid resize(
		const unsigned char *inImg,
		const int inWidth,
		const int inHeight,
		const int numComp,
		const int outWidth,
		const int outHeight,
		unsigned char *outImg
	) {
		const int pixelCount = outWidth * outHeight;
		const int pixelIdx = min(blockIdx.x * blockDim.x + threadIdx.x, pixelCount - 1);

		const float ratioW = float(outWidth) / inWidth;
		const float ratioH = float(outHeight) / inHeight;

		const int outX = pixelIdx % outWidth;
		const int outY = pixelIdx / outWidth;
		
		float2 sample;
		sample.x = (float(outX) + 0.5f) / ratioW;
		sample.y = (float(outY) + 0.5f) / ratioH;

		int2 floorSample = { int(floor(sample.x)), int(floor(sample.y)) };

		// TODO: these may depend on the sampling algorithm chosen.
		const int lancsozWindow = 3;
		int2 rangeX = {
			min(max(0, floorSample.x - lancsozWindow - 1), inWidth),
			min(max(0, floorSample.x + lancsozWindow + 1), inWidth)
		};
		int2 rangeY = {
			min(max(0, floorSample.y - lancsozWindow - 1), inHeight),
			min(max(0, floorSample.y + lancsozWindow + 1), inHeight)
		};

		convolve(inImg, lanczos2D, sample, rangeX, rangeY, inWidth, numComp, lancsozWindow, &outImg[pixelIdx * numComp]);
	}

}
