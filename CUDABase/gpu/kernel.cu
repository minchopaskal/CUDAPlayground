
#include <hip/hip_runtime.h>
// Includes that fix syntax highlighting
#ifdef CUDA_DEBUG
#include "device_launch_parameters.h"
#include "stdio.h"
#endif

extern "C" {

	__constant__ int arrSize;

	__global__ void adder(int *arrA, int *arrB, int *result) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		idx = min(idx, arrSize - 1);
		result[idx] = arrA[idx] + arrB[idx];
	}

}
